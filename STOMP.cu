#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <hip/hip_complex.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reverse.h>
#include <thrust/transform_scan.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <float.h>
#include <vector>
#include <unordered_map>
#include <math.h>

#include "hip/hip_runtime_api.h"
#include "STOMP.h"

using std::vector;
using std::unordered_map;
using std::make_pair;

static const unsigned int WORK_SIZE = 512;
static const unsigned int AMT_UNROLL = 4;

//This macro checks return value of the CUDA runtime call and exits
//the application if the call failed.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//This kernel computes a sliding mean with specified window size and a corresponding prefix sum array (A)
template<class DTYPE>
__global__ void sliding_mean(DTYPE* pref_sum,  size_t window, size_t size, DTYPE* means)
{
    const DTYPE coeff = 1.0 / (DTYPE) window;
    size_t a = blockIdx.x * blockDim.x + threadIdx.x;
    size_t b = blockIdx.x * blockDim.x + threadIdx.x + window;

    if(a == 0){
        means[a] = pref_sum[window - 1] * coeff;
    }
    if(a < size - 1){
        means[a + 1] = (pref_sum[b] - pref_sum[a]) * coeff;
    }
}

// This kernel computes the recipricol sliding standard deviaiton with specified window size, the corresponding means of each element, and the prefix squared sum at each element
// We actually compute the multiplicative inverse of the standard deviation, as this saves us from needing to do a division in the main kernel
template<class DTYPE>
__global__ void sliding_std(DTYPE* cumsumsqr, unsigned int window, unsigned int size, DTYPE* means, DTYPE* stds) {
    const DTYPE coeff = 1 / (DTYPE) window;
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    int b = blockIdx.x * blockDim.x + threadIdx.x + window;
    if (a == 0) {
        stds[a] = 1 / sqrt((cumsumsqr[window - 1] * coeff) - (means[a] * means[a]));
    }
    else if (b < size + window) {
        stds[a] = 1 / sqrt(((cumsumsqr[b - 1] - cumsumsqr[a - 1]) * coeff) - (means[a] * means[a]));
    }
}

template<class DTYPE>
void compute_statistics(const DTYPE *T, DTYPE *means, DTYPE *stds, size_t n, size_t m, hipStream_t s)
{
    square<DTYPE> sqr;
    dim3 grid(ceil(n / (double) WORK_SIZE), 1,1);
    dim3 block(WORK_SIZE, 1, 1);
    
    DTYPE *scratch;
    hipMalloc(&scratch, sizeof(DTYPE) * n);
    gpuErrchk(hipPeekAtLastError());
    
    thrust::device_ptr<const DTYPE> dev_ptr_T = thrust::device_pointer_cast(T);
    thrust::device_ptr<DTYPE> dev_ptr_scratch = thrust::device_pointer_cast(scratch);

    // Compute prefix sum in scratch
    thrust::inclusive_scan(thrust::cuda::par.on(s), dev_ptr_T, dev_ptr_T + n + m - 1, dev_ptr_scratch, thrust::plus<DTYPE>());
    gpuErrchk(hipPeekAtLastError());
    // Use prefix sum to compute sliding mean
    sliding_mean<DTYPE><<<grid, block, 0, s>>>(scratch, m, n, means);
    gpuErrchk(hipPeekAtLastError());
    // Compute prefix sum of squares in scratch
    thrust::transform_inclusive_scan(thrust::cuda::par.on(s), dev_ptr_T, dev_ptr_T + n + m - 1, dev_ptr_scratch, sqr,thrust::plus<DTYPE>());
    gpuErrchk(hipPeekAtLastError());
    // Use prefix sum of squares to compute the sliding standard deviation
    sliding_std<DTYPE><<<grid, block, 0, s>>>(scratch, m, n, means, stds);
    gpuErrchk(hipPeekAtLastError());
    hipStreamSynchronize(s);
    gpuErrchk(hipPeekAtLastError());
    hipFree(scratch);
    gpuErrchk(hipPeekAtLastError());
}

template<class DTYPE>
__global__ void elementwise_multiply_inplace(const DTYPE* A, DTYPE *B, const int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size) {
       B[tid] *= A[tid];
    }
} 

template<>
__global__ void elementwise_multiply_inplace(const hipDoubleComplex* A, hipDoubleComplex* B, const int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size) {
       B[tid] = hipCmul(A[tid], B[tid]);
    }
}

// A is input unaligned sliding dot products produced by ifft
// out is the computed vector of distances
template<class DTYPE>
__global__ void normalized_aligned_dot_products(const DTYPE* A, const DTYPE divisor,
                                                const unsigned int m, const unsigned int n,
                                                DTYPE* QT)
{
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    if (a < n) {
        QT[a] = A[a + m - 1] / divisor;
    }
}

template<class DTYPE>
__global__ void populate_reverse_pad(const DTYPE *Q, DTYPE *Q_reverse_pad, const int window_size, const int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < window_size) {
        Q_reverse_pad[tid] = Q[window_size - 1 - tid];
    }else if(tid < size){ 
        Q_reverse_pad[tid] = 0;
    }
}

template<class DTYPE, class CUFFT_DTYPE>
void sliding_dot_products_and_distance_profile(DTYPE* T, DTYPE* Q, DTYPE *QT, const int size, const int window_len, hipStream_t s)
{        

    const int n = size - window_len + 1;
    const int cufft_data_size = size / 2 + 1;
    dim3 grid(ceil(n / (float) WORK_SIZE), 1, 1);
    dim3 block(WORK_SIZE, 1, 1);

    hipfftHandle fft_plan, ifft_plan;    
    DTYPE *Q_reverse_pad;
    CUFFT_DTYPE *Tc, *Qc;
    hipfftPlan1d(&fft_plan, size, CUFFT_FORWARD_PLAN, 1);
    hipfftPlan1d(&ifft_plan, size, CUFFT_REVERSE_PLAN, 1);
    hipfftSetStream(fft_plan, s);
    hipfftSetStream(ifft_plan,s);
    hipMalloc(&Q_reverse_pad, sizeof(DTYPE) * size);
    hipMalloc(&Tc, sizeof(CUFFT_DTYPE) * cufft_data_size);
    hipMalloc(&Qc, sizeof(CUFFT_DTYPE) * cufft_data_size);
    
    // Compute the FFT of the time series
    CUFFT_FORWARD__(fft_plan, T, Tc);
    gpuErrchk(hipPeekAtLastError());

    // Reverse and zero pad the query
    populate_reverse_pad<DTYPE><<<dim3(ceil(size / (float) WORK_SIZE),1,1), block, 0, s>>>(Q, Q_reverse_pad, window_len, size);
    gpuErrchk(hipPeekAtLastError());
    
    // Compute the FFT of the query
    CUFFT_FORWARD__(fft_plan, Q_reverse_pad, Qc);
    gpuErrchk(hipPeekAtLastError());
    
    elementwise_multiply_inplace<<<dim3(ceil(cufft_data_size / (float) WORK_SIZE), 1, 1), block, 0, s>>>(Tc, Qc, cufft_data_size);
    gpuErrchk(hipPeekAtLastError());

    // Compute the ifft
    // Use the space for the query as scratch space as we no longer need it
    CUFFT_REVERSE__(ifft_plan, Qc, Q_reverse_pad);
    gpuErrchk(hipPeekAtLastError());
    
    normalized_aligned_dot_products<DTYPE><<<grid, block, 0, s>>>(Q_reverse_pad, size, window_len, n, QT);
    gpuErrchk(hipPeekAtLastError());
    
    hipFree(Q_reverse_pad);
    hipFree(Tc);
    hipFree(Qc);
    hipfftDestroy(fft_plan);
    hipfftDestroy(ifft_plan);
} 





//Atomically updates the MP/idxs using a single 64-bit integer. We lose a small amount of precision in the output, if we do not do this we are unable
// to atomically update both the matrix profile and the indexes without using a critical section and dedicated locks.
__device__ inline void MPatomicMax(volatile unsigned long long int* address, float val, unsigned int idx)
{
    mp_entry loc, loctest;
    loc.floats[0] = val;
    loc.ints[1] = idx;
    loctest.ulong = *address;
    while (loctest.floats[0] < val){
        loctest.ulong = atomicCAS((unsigned long long int*) address, loctest.ulong,  loc.ulong);
    }
}

template<unsigned int tile_height>
//Updates the global matrix profile based on a block-local, cached version
__device__ inline void UpdateMPGlobalMax(unsigned long long* profile, volatile mp_entry* localMP, const int chunk, const int offset, const int n){
    
    int x = chunk*(tile_height)+threadIdx.x;
    if(x < n && ((mp_entry*) profile)[x].floats[0] < localMP[threadIdx.x+offset].floats[0])
    {
            MPatomicMax(&profile[x], localMP[threadIdx.x+offset].floats[0], localMP[threadIdx.x+offset].ints[1]);
    }
}

template<class DTYPE, unsigned int BLOCKSZ, unsigned int tile_height>
__device__ inline void initialize_tile_memory(const unsigned long long int *profile, const double *T,
                                              const double *means, const double *inv_stds,
                                              volatile mp_entry localMPMain[], volatile mp_entry localMPOther[],
                                              DTYPE A_low[], DTYPE A_high[], DTYPE B_low[], DTYPE B_high[],
                                              DTYPE mean_x[], DTYPE mean_y[], DTYPE inv_std_x[],
                                              DTYPE inv_std_y[], const unsigned int n, const unsigned int m,
                                              const unsigned int mainStart, const unsigned int otherStart,
                                              const unsigned int x, const unsigned int y)
{
    // Update local cache to point to the next chunk of the MP
    // We may not get the 'freshest' values from the global array, but it doesn't really matter too much
    if (mainStart + threadIdx.x < n) {
        localMPMain[threadIdx.x].ulong = profile[mainStart + threadIdx.x];
    } else {
        localMPMain[threadIdx.x].floats[0] = CC_MIN;
        localMPMain[threadIdx.x].ints[1] = 0;
    }

    // Each thread grabs 2 values for the main cache
    if (threadIdx.x < tile_height && mainStart+threadIdx.x+BLOCKSZ < n) {
        localMPMain[BLOCKSZ + threadIdx.x].ulong = profile[mainStart + BLOCKSZ + threadIdx.x];
    } else if (threadIdx.x < tile_height) {
        localMPMain[threadIdx.x + BLOCKSZ].floats[0] = CC_MIN;
        localMPMain[threadIdx.x + BLOCKSZ].ints[1] = 0;
    }
    
    // We also update the cache for the transposed tile
    if (threadIdx.x < tile_height && otherStart+threadIdx.x < n) {
        localMPOther[threadIdx.x].ulong = profile[otherStart + threadIdx.x];
    } else if (threadIdx.x < tile_height) {
        localMPOther[threadIdx.x].floats[0] = CC_MIN;
        localMPOther[threadIdx.x].ints[1] = 0;
    }

    // Update the other cached values to reflect the upcoming tile
    if (x <  n + m - 1) {
        A_low[threadIdx.x] = T[x];
    }
    if (threadIdx.x < tile_height && x + BLOCKSZ < n + m - 1) {
        A_low[threadIdx.x + BLOCKSZ] = T[x + BLOCKSZ];
    }
    
    if (x + m < n + m - 1) {
        A_high[threadIdx.x] = T[x + m];
    }
    if (threadIdx.x < tile_height && x + BLOCKSZ + m < n + m - 1) {
        A_high[threadIdx.x + BLOCKSZ] = T[x + BLOCKSZ + m];
    }
    if (threadIdx.x < tile_height && y + threadIdx.x < n + m - 1) {
        B_low[threadIdx.x] = T[y + threadIdx.x];
    }
    if (threadIdx.x < tile_height && y + threadIdx.x + m < n + m - 1) {
        B_high[threadIdx.x] = T[y + threadIdx.x + m];
    }
    if (x < n) {
        inv_std_x[threadIdx.x] = inv_stds[x];
        // We precompute part of the distance calculation in the mean_x variable
        // This saves us a multiply in the main loop
        mean_x[threadIdx.x] = means[x] * m;
    }
    if (threadIdx.x < tile_height && x + BLOCKSZ < n) {
        inv_std_x[threadIdx.x + BLOCKSZ] = inv_stds[x + BLOCKSZ];
        // We precompute part of the distance calculation in the mean_x variable
        // This saves us a multiply in the main loop
        mean_x[threadIdx.x + BLOCKSZ] = means[x + BLOCKSZ] * m;
    }
    if (threadIdx.x < tile_height && y + threadIdx.x < n) {
        inv_std_y[threadIdx.x] = inv_stds[y + threadIdx.x];
        mean_y[threadIdx.x] = means[y + threadIdx.x];
    }
}

//Computes the matrix profile given the sliding dot products for the first query and the precomputed data statisics
template<class DTYPE, unsigned int BLOCKSZ, unsigned int UNROLL_COUNT>
__global__ void WavefrontUpdateSelfJoinMaxSharedMem(const double* QT, const double* T, const double* inv_stds, const double* means, unsigned long long int* profile, unsigned int m, unsigned int n, int startPos, int numDevices, struct reg_mem<UNROLL_COUNT> mem)
{
    //Factor and threads per block must both be powers of two where: factor <= threads per block
    // UNROLL_COUNT * factor must also evenly divide WORK_SIZE
    // The values that are set here should give good performance already
    // but may be fine tuned for other Nvidia architectures
    //Use the smallest power of 2 possible for your GPU
    const int factor = 4;
    const int tile_height = BLOCKSZ / factor;
    const int tile_width = tile_height + BLOCKSZ;
    __shared__ mp_entry localMPMain[tile_width];
    __shared__ mp_entry localMPOther[tile_height];
    __shared__ DTYPE A_low[tile_width];
    __shared__ DTYPE A_high[tile_width];
    __shared__ DTYPE inv_std_x[tile_width];
    __shared__ DTYPE inv_std_y[tile_height];
    __shared__ DTYPE mean_x[tile_width];
    __shared__ DTYPE mean_y[tile_height];
    __shared__ DTYPE B_high[tile_height];
    __shared__ DTYPE B_low[tile_height];


    int exclusion = (m / 4);
    int a = ((blockIdx.x * numDevices) + startPos) * BLOCKSZ + threadIdx.x;
    double qt_curr;
    int localX = threadIdx.x;
    int localY = 0;
    int chunkIdxMain = (a / BLOCKSZ) * factor;
    int chunkIdxOther = 0;
    int mainStart = tile_height * chunkIdxMain;
    int otherStart = 0;
    if (a < n) {
        mem.qt[0] = QT[a];
    }
    
    // x is the global column of the distance matrix
    // y is the global row of the distance matrix
    // Each thread starts on the first row and works its way to the down-right diagonal
    int x = a;
    int y = 0;

    // The first threads are acutally computing the trivial match between the same subsequence
    // we exclude these from the calculation
    bool excluded;
    if (x <= exclusion) {
        excluded = true;
    } else {
        excluded = false;
    }
    // Initialize the first tile's shared memory 
    initialize_tile_memory<DTYPE, BLOCKSZ, tile_height>(profile, T, means, inv_stds, localMPMain, localMPOther,
                                            A_low, A_high, B_low, B_high, mean_x, mean_y, inv_std_x,
                                            inv_std_y, n, m, mainStart, otherStart, x, y);

    /////////////////////////////////////    
    // Main loop
    /////////////////////////////////////
    // Each threadblock finds all the distances on a 'metadiagonal'
    // We use a tiled approach for each thread block
    // The tiles are horizontal slices of the diagonal, think of a parallelogram cut
    // from a diagonal slice of the distance matrix 
    while (mainStart < n && otherStart < n)
    {
        // Start of new tile, sync
        __syncthreads();

        // The first 'm/4' diagonals are computing the exclusion zone, so we don't want to
        // include their distances in the calculation
        if (excluded) {
                x += tile_height;
                y += tile_height;
        } else {
            // Process the tile
            // Each iteration generates the next 4 distances
            // This loop is partially unrolled to improve instruction level parallelism
            while (x < n - UNROLL_COUNT + 1 && localY < tile_height)
            {
                // Update the QT value for the next iteration
                #pragma unroll
                for (int i = 0; i < UNROLL_COUNT - 1; ++i) {
                    mem.qt[i + 1] = mem.qt[i] - A_low[localX + i] * B_low[localY + i] + A_high[localX + i] * B_high[localY + i];
                }
                
                // Compute the next partial distance value
                // We defer some of the calculation until after the kernel has finished, this saves us several
                // long latency math operations in this critical path.
                // The distance computed here can be converted to the true z-normalized euclidan
                // distance in constant time
                // mean_x has already been multiplied with the window size 'm' when the tile was populated
                #pragma unroll
                for (int i = 0; i < UNROLL_COUNT; ++i) {
                    mem.dist[i] = (static_cast<float>(mem.qt[i]) - (mean_x[localX + i] * mean_y[localY + i])) * inv_std_x[localX + i] * inv_std_y[localY + i];
                }


                mem.qt[0] = mem.qt[UNROLL_COUNT - 1] - A_low[localX + UNROLL_COUNT - 1] * B_low[localY + UNROLL_COUNT - 1] + A_high[localX + UNROLL_COUNT - 1] * B_high[localY + UNROLL_COUNT - 1];

                // Update the cache with the new max value atomically
                #pragma unroll
                for (int i = 0; i < UNROLL_COUNT; ++i) {
                    MPatomicMax((unsigned long long int*) (localMPMain + localX + i), mem.dist[i], y + i);
                    MPatomicMax((unsigned long long int*) (localMPOther + localY + i), mem.dist[i], x + i);
                }

                x += UNROLL_COUNT;
                y += UNROLL_COUNT;
                localX += UNROLL_COUNT;
                localY += UNROLL_COUNT;
            }

            qt_curr = mem.qt[0];

            // Finish the remaining iterations of the tile
            while (x < n && localY < tile_height) {
                float dist = (static_cast<float>(qt_curr) - (mean_x[localX] * mean_y[localY])) * inv_std_x[localX] * inv_std_y[localY];
                qt_curr = qt_curr - A_low[localX] * B_low[localY] + A_high[localX] * B_high[localY];
                MPatomicMax((unsigned long long int*) (localMPMain + localX), dist, y);
                MPatomicMax((unsigned long long int*) (localMPOther + localY), dist, x);

                x++;
                y++;
                localX++;
                localY++;
            }

        }

        // After this sync, the caches will be updated with the best so far values for this tile
        __syncthreads();

        // If we updated any values in the cached MP, try to push them to the global "master" MP
        UpdateMPGlobalMax<tile_height>(profile, localMPMain, chunkIdxMain, 0, n);
        if (threadIdx.x < tile_height) {
            UpdateMPGlobalMax<tile_height>(profile, localMPMain, chunkIdxMain + factor, BLOCKSZ, n);
            UpdateMPGlobalMax<tile_height>(profile, localMPOther, chunkIdxOther, 0, n);
        }


        // Update the tile position
        mainStart += tile_height;
        otherStart += tile_height;

        // Make sure our updates were committed before we pull in the next tile
        __threadfence_block();
    
        // Initialize the next tile's shared memory 
        initialize_tile_memory<DTYPE, BLOCKSZ, tile_height>(profile, T, means, inv_stds, localMPMain, localMPOther,
                                                A_low, A_high, B_low, B_high, mean_x, mean_y, inv_std_x,
                                                inv_std_y, n, m, mainStart, otherStart, x, y);

        // Reset the tile local positions
        localY = 0;
        localX = threadIdx.x;
        chunkIdxMain++;
        chunkIdxOther++;    
    }
}

__global__ void cross_correlation_to_ed(float *profile, unsigned int n, unsigned int m) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) {
        profile[tid] = sqrt(max(2*(m - profile[tid]), 0.0));
    }
}

// The STOMP algorithm
template<class DTYPE, class CUFFT_DTYPE>
void do_STOMP(const vector<DTYPE> &T_h, vector<float> &profile_h, vector<unsigned int> &profile_idx_h, const unsigned int m, const vector<int> &devices) {
    if(devices.empty()) {
        printf("Error: no gpu provided\n");
        exit(0);
    }
    
    size_t n = T_h.size() - m + 1;
    
    unordered_map<int, DTYPE*> T_dev, QT_dev, means, stds;
    unordered_map<int, float*> profile_dev;
    unordered_map<int, unsigned long long int*> profile_merged;
    unordered_map<int, unsigned int*> profile_idx_dev;
    unordered_map<int, hipEvent_t> clocks_start, clocks_end;
    unordered_map<int, hipStream_t> streams;

    // Allocate and initialize memory
    for (auto device : devices) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        T_dev.insert(make_pair(device, (DTYPE*) 0));
        QT_dev.insert(make_pair(device, (DTYPE*) 0));
        means.insert(make_pair(device, (DTYPE*) 0));
        stds.insert(make_pair(device, (DTYPE*) 0));
        profile_dev.insert(make_pair(device,(float*) NULL));
        profile_merged.insert(make_pair(device,(unsigned long long int*) NULL));
        profile_idx_dev.insert(make_pair(device,(unsigned int *) NULL));


        hipMalloc(&T_dev.at(device), sizeof(DTYPE) * T_h.size());
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_dev.at(device), sizeof(float) * profile_h.size());
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_idx_dev.at(device), sizeof(unsigned int) * profile_idx_h.size());
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&QT_dev.at(device), sizeof(DTYPE) * profile_h.size());
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&means.at(device), sizeof(DTYPE) * profile_h.size());
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&stds.at(device), sizeof(DTYPE) * profile_h.size());
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_merged.at(device), sizeof(unsigned long long int) * n);
        gpuErrchk(hipPeekAtLastError());
        hipEvent_t st, ed;
        hipEventCreate(&ed);
        gpuErrchk(hipPeekAtLastError());
        hipEventCreate(&st);
        gpuErrchk(hipPeekAtLastError());
        clocks_start.emplace(device, st);
        clocks_end.emplace(device, ed);
        hipStream_t s;
        hipStreamCreate(&s);
        gpuErrchk(hipPeekAtLastError());
        streams.emplace(device, s);
    }

    MPIDXCombine combiner;
    int num_workers = ceil(n / (float) devices.size());
    
    // Asynchronously copy relevant data, precompute statistics, generate partial matrix profile
    int count = 0;
    for (auto &device : devices) {
        hipSetDevice(device);
        hipMemcpyAsync(T_dev[device], T_h.data(), sizeof(DTYPE) * T_h.size(), hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profile_dev[device], profile_h.data(), sizeof(float) * profile_h.size(), hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profile_idx_dev[device], profile_idx_h.data(), sizeof(unsigned int) * profile_idx_h.size(), hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());

        // Computing the statistics for each device is overkill, but it avoids needing to do some staging on the host if P2P transfer doesn't work
        compute_statistics<DTYPE>(T_dev[device], means[device], stds[device], n, m, streams.at(device));
        sliding_dot_products_and_distance_profile<DTYPE, CUFFT_DTYPE>(T_dev[device], T_dev[device], QT_dev[device], T_h.size(), m, streams.at(device));
        
        thrust::device_ptr<unsigned long long int> ptr = thrust::device_pointer_cast(profile_merged[device]);
        thrust::transform(thrust::cuda::par.on(streams.at(device)), profile_dev[device], profile_dev[device] + n, profile_idx_dev[device], profile_merged[device], combiner);
        printf("Start main kernel on GPU %d\n", device);
        //hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
        hipEventRecord(clocks_start[device], streams.at(device));
        WavefrontUpdateSelfJoinMaxSharedMem<float, WORK_SIZE, AMT_UNROLL><<<dim3(ceil(num_workers / (double) WORK_SIZE), 1, 1),dim3(WORK_SIZE, 1,1), 0, streams.at(device)>>>(QT_dev[device], T_dev[device], stds[device], means[device], profile_merged[device], m, n, count, devices.size(), reg_mem<AMT_UNROLL>());
        hipEventRecord(clocks_end[device], streams.at(device));
        ++count;
    }
   
    float time;
    for(auto &device : devices) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        hipStreamSynchronize(streams.at(device));
        hipEventElapsedTime(&time, clocks_start[device], clocks_end[device]);
        gpuErrchk(hipPeekAtLastError());
        hipEventDestroy(clocks_start.at(device));
        hipEventDestroy(clocks_end.at(device));
        printf("Device %d took %f seconds\n", device, time / 1000);
    }

    printf("Finished STOMP to generate partial matrix profile of size %lu on %d devices:\n", n, devices.size());

    // Free unneeded resources
    for (auto &device : devices) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        hipFree(T_dev[device]);
        gpuErrchk(hipPeekAtLastError());
        // Keep the profile for the first device as a staging area for the final result
        if (device != devices.at(0)) { 
            hipFree(profile_dev[device]);
            gpuErrchk(hipPeekAtLastError());
            hipFree(profile_idx_dev[device]);
            gpuErrchk(hipPeekAtLastError());
        }
        hipFree(QT_dev[device]);
        gpuErrchk(hipPeekAtLastError());
        hipFree(means[device]);
        gpuErrchk(hipPeekAtLastError());
        hipFree(stds[device]);
        gpuErrchk(hipPeekAtLastError());
        hipStreamDestroy(streams.at(device));
        gpuErrchk(hipPeekAtLastError());
    }
   

    // Consolidate the partial matrix profiles to a single vector using the first gpu 
    printf("Merging partial matrix profiles into final result\n");
    vector<unsigned long long int> partial_profile_host(n);
    hipSetDevice(devices.at(0));
    gpuErrchk(hipPeekAtLastError());
    auto ptr_profile = thrust::device_ptr<float>(profile_dev[devices.at(0)]);
    auto ptr_index = thrust::device_ptr<unsigned int>(profile_idx_dev[devices.at(0)]);
    auto ptr_merged = thrust::device_ptr<unsigned long long int>(profile_merged[devices.at(0)]);
    auto iter_begin = thrust::make_zip_iterator(thrust::make_tuple(ptr_profile, ptr_index, ptr_merged));
    auto iter_end = thrust::make_zip_iterator(thrust::make_tuple(ptr_profile + n, ptr_index + n, ptr_merged + n));
    for(int i = 0; i < devices.size(); ++i) {
        hipSetDevice(devices.at(i));
        gpuErrchk(hipPeekAtLastError());
        if (i != 0) {
            hipMemcpy(partial_profile_host.data(), profile_merged[devices.at(i)], n * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
            gpuErrchk(hipPeekAtLastError());
            hipFree(profile_merged[devices.at(i)]);
            gpuErrchk(hipPeekAtLastError());
            hipSetDevice(devices.at(0));
            gpuErrchk(hipPeekAtLastError());
            hipMemcpy(profile_merged[0], partial_profile_host.data(), n * sizeof(unsigned long long int), hipMemcpyHostToDevice);
            gpuErrchk(hipPeekAtLastError());
        }
        thrust::for_each(iter_begin, iter_end, max_with_index());
        gpuErrchk(hipPeekAtLastError());
    }
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    hipSetDevice(devices.at(0));
    gpuErrchk(hipPeekAtLastError());
         
    // Compute the final distance calculation to convert cross correlation computed earlier into euclidean distance
    cross_correlation_to_ed<<<dim3(ceil(n / (float) WORK_SIZE), 1, 1), dim3(WORK_SIZE, 1, 1)>>>(profile_dev[devices.at(0)], n, m); 
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(profile_idx_h.data(), profile_idx_dev[devices.at(0)], sizeof(unsigned int) * n, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(profile_h.data(), profile_dev[devices.at(0)], sizeof(float) * n, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    hipFree(profile_idx_dev[devices.at(0)]);
    gpuErrchk(hipPeekAtLastError());
    hipFree(profile_dev[devices.at(0)]);
    gpuErrchk(hipPeekAtLastError());
    hipFree(profile_merged[devices.at(0)]);
    gpuErrchk(hipPeekAtLastError());

}

//Reads input time series from file
template<class DTYPE>
void readFile(const char* filename, vector<DTYPE>& v, const char *format_str) 
{
    FILE* f = fopen( filename, "r");
    if(f == NULL){
        printf("Unable to open %s for reading, please make sure it exists\n", filename);
        exit(0);
    }
    DTYPE num;
    while(!feof(f)){
            fscanf(f, format_str, &num);
            v.push_back(num);
        }
    v.pop_back();
    fclose(f);
}
    


int main(int argc, char** argv) {

    if(argc < 5) {
        printf("Usage: STOMP <window_len> <input file> <profile output file> <index output file> [Optional: list of GPU device numbers to run on]\n");
        exit(0);
    }

    int window_size = atoi(argv[1]);
    
    vector<double> T_h;
    readFile<double>(argv[2], T_h, "%lf");
    int n = T_h.size() - window_size + 1;
    vector<float> profile(n, CC_MIN);
    vector<unsigned int> profile_idx(n, 0);
    
    hipFree(0);
    
    vector<int> devices;
    
    if(argc == 5) {
        // Use all available devices 
        int num_dev;
        hipGetDeviceCount(&num_dev);
        for(int i = 0; i < num_dev; ++i){ 
            devices.push_back(i);
        }
    } else {
        // Use the devices specified
        int x = 5;
        while (x < argc) {
            devices.push_back(atoi(argv[x]));
            ++x;
        }
    }
    
    printf("Starting STOMP\n");
     
    do_STOMP<double, hipDoubleComplex>(T_h, profile, profile_idx, window_size, devices);
    
    printf("Now writing result to files\n");
    FILE* f1 = fopen( argv[3], "w");
    FILE* f2 = fopen( argv[4], "w");
    for(int i = 0; i < profile.size(); ++i){
        fprintf(f1, "%f\n", profile[i]);
        fprintf(f2, "%u\n", profile_idx[i] + 1);
    }
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipDeviceReset());
    fclose(f1);
    fclose(f2);
    printf("Done\n");
    return 0;
}



